#include "hip/hip_runtime.h"
/* 
 /*Render program functions, modified from work by NVIDIA
 */

/* Copyright (c) 2016, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include "RenderFuncs.h"
#include <optixu/optixu_aabb.h>
#include "random.h"

#define SAMPLE_ITERS 4.f

#define SAMPLE_ITERS_RECIP (1.f / SAMPLE_ITERS)

rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, ); 
rtDeclareVariable(float3, shading_normal,   attribute shading_normal, ); 

rtDeclareVariable(PerRayData_radiance, prd_radiance, rtPayload, );
rtDeclareVariable(PerRayData_shadow,   prd_shadow,   rtPayload, );

rtDeclareVariable(optix::Ray, ray,          rtCurrentRay, );
rtDeclareVariable(float,      t_hit,        rtIntersectionDistance, );
rtDeclareVariable(uint2,      launch_index, rtLaunchIndex, );

rtDeclareVariable(unsigned int, radiance_ray_type, , );
rtDeclareVariable(unsigned int, shadow_ray_type , , );
rtDeclareVariable(float,        scene_epsilon, , );
rtDeclareVariable(rtObject,     top_object, , );

//
// Pinhole camera implementation
//
rtDeclareVariable(float3,        eye, , );
rtDeclareVariable(float3,        U, , );
rtDeclareVariable(float3,        V, , );
rtDeclareVariable(float3,        W, , );
rtDeclareVariable(float3,        bad_color, , );
rtBuffer<uchar4, 2>              output_buffer;

RT_PROGRAM void pinhole_camera()
{
  size_t2 screen = output_buffer.size();

  float2 d = make_float2(launch_index) / make_float2(screen) * 2.f - 1.f;
  float3 ray_origin = eye;
  float3 ray_direction = normalize(d.x*U + d.y*V + W);

  optix::Ray ray(ray_origin, ray_direction, radiance_ray_type, scene_epsilon);

  PerRayData_radiance prd;
  prd.importance = 1.f;
  prd.depth = 0;

  rtTrace(top_object, ray, prd);

  output_buffer[launch_index] = make_color( prd.result );
}

/*
	Thin lens camera, to produce simple focal plane/depth of field effect. Based
	primarily on implementation shown in Physically Based Rendering by Pharr, 
	Jakob, and Humphreys

*/

rtDeclareVariable(float, f_length, , );
rtDeclareVariable(float, lens_rad, , );

//rtDeclareVariable(float, dist, , );

RT_PROGRAM void thin_lens_camera()
{
	// Get ray direction of eye to image plane in the same way as before
	size_t2 screen = output_buffer.size();

	float2 d = make_float2(launch_index) / make_float2(screen) * 2.f - 1.f;
	float3 ray_origin = eye;
	float3 ray_direction = normalize(d.x*U + d.y*V + W);

	// If lens_radius is 0, treat as pinhole
	if(lens_rad == 0)
	{
		optix::Ray ray(ray_origin, ray_direction, radiance_ray_type, scene_epsilon);

		PerRayData_radiance prd;
		prd.importance = 1.f;
		prd.depth = 0;

		rtTrace(top_object, ray, prd);

		output_buffer[launch_index] = make_color(prd.result);
	
	}
	else
	{
		///*

		//	Any ray that enters parallel to the axis on one side of the lens proceeds
		//	towards the focal point f on the other side

		//	Any ray that arrives at the lens after passing through the focal point on
		//	the front side comes out paralle to taxis on other side

		//	Any ray that passes through center of lens will not change direction

		//	Relation between distance s and image distance D' (Thin Lens Formula):

		//	(1/D) + (1/D') = (1/f);

		//	-> (1/D') = (1/f) - (1/D);

		//	-> D' = 1/((1/f) - (1/D));

		//	-> D' = (f * D) / (f + D)
		//*/

		//// Get D' value from distance and focal length
		//float dist_prime = (f_length * dist) / (f_length + dist);


		float3 result_color = make_float3(0.f, 0.f, 0.f);

		for (int i = 0; i < SAMPLE_ITERS; i++)
		{
			// 1) Sample point on lens
			unsigned seed = tea<2>(d.x, d.y);
			float2 sample_point = make_float2(rnd(seed) - 0.5f, rnd(seed) - 0.5f);

			//		float2 sample_point = d;

			//float2 sample_point = make_float2((d.x + 1.f) * 0.5f, (d.y + 1.f) * 0.5f);


			float2 disc_point =
				concentric_sample_disk(sample_point);


			float2 p_lens = lens_rad * disc_point;


			// 2) Compute point on plane of focus

			float ft = f_length / ray_direction.z;

			float3 pFocus = ray_origin + ray_direction * ft;

			// 3) Update ray for effect on lens

			ray_origin = make_float3(p_lens.x, p_lens.y, 0.f);
			ray_direction = normalize(pFocus - ray_origin);

			optix::Ray ray(ray_origin, ray_direction, radiance_ray_type, scene_epsilon);


			PerRayData_radiance prd;
			prd.importance = 1.f;
			prd.depth = 0;

			rtTrace(top_object, ray, prd);
			result_color += prd.result;
		}
		result_color *= SAMPLE_ITERS_RECIP;

		output_buffer[launch_index] = make_color(result_color);//make_color(prd.result);
	}

}


//
// Returns solid color for miss rays
//

rtDeclareVariable(float3, bg_color, , );
RT_PROGRAM void miss()
{
	prd_radiance.result = bg_color;
}

//
// Terminates and fully attenuates ray after any hit
//
RT_PROGRAM void any_hit_shadow()
{
	// this material is opaque, so it fully attenuates all shadow rays
	prd_shadow.attenuation = make_float3(0);

	rtTerminateRay();
}

//
// Attenuates shadow rays for shadowing transparent objects
//

rtDeclareVariable(float3, shadow_attenuation, , );

RT_PROGRAM void glass_any_hit_shadow()
{
  float3 world_normal = 
	  normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, shading_normal ) );
  float nDi = fabs(dot(world_normal, ray.direction));

  prd_shadow.attenuation *=
	  1-fresnel_schlick(nDi, 5, 1-shadow_attenuation, make_float3(1));

  rtIgnoreIntersection();
}


//
// Dielectric surface shader
//
rtDeclareVariable(float3,       cutoff_color, , );
rtDeclareVariable(float,        fresnel_exponent, , );
rtDeclareVariable(float,        fresnel_minimum, , );
rtDeclareVariable(float,        fresnel_maximum, , );
rtDeclareVariable(float,        refraction_index, , );
rtDeclareVariable(int,          refraction_maxdepth, , );
rtDeclareVariable(int,          reflection_maxdepth, , );
rtDeclareVariable(float3,       refraction_color, , );
rtDeclareVariable(float3,       reflection_color, , );
rtDeclareVariable(float3,       extinction_constant, , );
rtDeclareVariable(float, importance_cutoff, , );
rtDeclareVariable(int, max_depth, , );

RT_PROGRAM void glass_closest_hit_radiance()
{
  // intersection vectors
  const float3 h = ray.origin + t_hit * ray.direction;            // hitpoint
  const float3 n = 
	  normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal)); // normal
  const float3 i = ray.direction;                           // incident direction

  float reflection = 1.0f;
  float3 result = make_float3(0.0f);

  float3 beer_attenuation;
  if(dot(n, ray.direction) > 0){
    // Beer's law attenuation
    beer_attenuation = exp(extinction_constant * t_hit);
  } else {
    beer_attenuation = make_float3(1);
  }

  // refraction
  if (prd_radiance.depth < min(refraction_maxdepth, max_depth))
  {
    float3 t;											// transmission direction
    if ( refract(t, i, n, refraction_index) )
    {

      // check for external or internal reflection
      float cos_theta = dot(i, n);
      if (cos_theta < 0.0f)
        cos_theta = -cos_theta;
      else
        cos_theta = dot(t, n);

      reflection = fresnel_schlick(cos_theta,
		  fresnel_exponent,
		  fresnel_minimum,
		  fresnel_maximum);

      float importance = 
		  prd_radiance.importance 
		  * (1.0f-reflection) 
		  * optix::luminance( refraction_color * beer_attenuation );
      if ( importance > importance_cutoff ) {
        optix::Ray ray( h, t, radiance_ray_type, scene_epsilon );
        PerRayData_radiance refr_prd;
        refr_prd.depth = prd_radiance.depth+1;
        refr_prd.importance = importance;

        rtTrace( top_object, ray, refr_prd );
        result += (1.0f - reflection) * refraction_color * refr_prd.result;
      } else {
        result += (1.0f - reflection) * refraction_color * cutoff_color;
      }
    }
    // else TIR
  }

  // reflection
  if (prd_radiance.depth < min(reflection_maxdepth, max_depth))
  {
    float3 r = reflect(i, n);

    float importance = prd_radiance.importance 
		* reflection 
		* optix::luminance( reflection_color * beer_attenuation );
    if ( importance > importance_cutoff ) {
      optix::Ray ray( h, r, radiance_ray_type, scene_epsilon );
      PerRayData_radiance refl_prd;
      refl_prd.depth = prd_radiance.depth+1;
      refl_prd.importance = importance;

      rtTrace( top_object, ray, refl_prd );
      result += reflection * reflection_color * refl_prd.result;
    } else {
      result += reflection * reflection_color * cutoff_color;
    }
  }

  result = result * beer_attenuation;

  prd_radiance.result = result;
}


//
// (UPDATED)
// Phong surface shading with shadows 
//

rtDeclareVariable(float3, Ka, , );
rtDeclareVariable(float3, Ks, , );
rtDeclareVariable(float, phong_exp, , );
rtDeclareVariable(float3, Kd, , );
rtDeclareVariable(float3, ambient_light_color, , );
rtBuffer<BasicLight>        lights;
rtDeclareVariable(rtObject, top_shadower, , );

RT_PROGRAM void closest_hit_radiance3()
{
	float3 world_geo_normal = 
		normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, geometric_normal));
	float3 world_shade_normal = 
		normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal));
	float3 ffnormal = 
		faceforward(world_shade_normal, -ray.direction, world_geo_normal);
	float3 color = Ka * ambient_light_color;

	float3 hit_point = ray.origin + t_hit * ray.direction;

	for (int i = 0; i < lights.size(); ++i) {
		BasicLight light = lights[i];
		float3 L = normalize(light.pos - hit_point);
		float nDl = dot(ffnormal, L);

		if (nDl > 0.0f) {
			// cast shadow ray
			PerRayData_shadow shadow_prd;
			shadow_prd.attenuation = make_float3(1.0f);
			float Ldist = length(light.pos - hit_point);
			optix::Ray shadow_ray(hit_point,
				L,
				shadow_ray_type,
				scene_epsilon,
				Ldist);
			rtTrace(top_shadower, shadow_ray, shadow_prd);
			float3 light_attenuation = shadow_prd.attenuation;

			if (fmaxf(light_attenuation) > 0.0f) {
				float3 Lc = light.color * light_attenuation;
				color += Kd * nDl * Lc;

				float3 H = normalize(L - ray.direction);
				float nDh = dot(ffnormal, H);
				if (nDh > 0)
					color += Ks * Lc * pow(nDh, phong_exp);
			}

		}
	}
	prd_radiance.result = color;
}


//
// Set pixel to solid color upon failure
//
RT_PROGRAM void exception()
{
  output_buffer[launch_index] = make_color( bad_color );
}
